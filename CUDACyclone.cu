#include "hip/hip_runtime.h"
// Build:
// RTX 30xx : nvcc -O3 --use_fast_math -std=c++17 -arch=sm_86 -Xptxas=-O3,-dlcm=ca -maxrregcount=64 -Xcompiler -pthread CUDACyclone.cu -o CUDACyclone
// RTX 50xx : nvcc -O3 --use_fast_math -std=c++17 -arch=sm_90 -Xptxas=-O3,-dlcm=ca -maxrregcount=64 -Xcompiler -pthread CUDACyclone.cu -o CUDACyclone
// (or use make)

// =================== includes ===================
#include <hip/hip_runtime.h>
#include <>
#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <iomanip>
#include <sstream>
#include <string>
#include <thread>
#include <chrono>
#include <cmath>
#include <vector>

#include "CUDAMath.h"
#include "sha256.h"
#include "CUDAHash.cuh"
#include "CUDAUtils.h"
#include "CUDAStructures.h"

// ================= email config =================
static const char* EMAIL_TO      = "email@to.com";
static const char* EMAIL_FROM    = "email@from.com";
static const char* EMAIL_SUBJECT = "CUDACyclone: result found";

// ================= email helper =================
static bool send_email_msmtp(const std::string& to,
                             const std::string& from,
                             const std::string& subject,
                             const std::string& html)
{
    if (to.empty() || from.empty() || subject.empty() || html.empty()) return false;

    std::ostringstream mail;
    mail << "From: " << from << "\n"
         << "To: " << to << "\n"
         << "Subject: " << subject << "\n"
         << "MIME-Version: 1.0\n"
         << "Content-Type: text/html; charset=UTF-8\n"
         << "Content-Transfer-Encoding: 8bit\n"
         << "\n" << html << "\n";

#if defined(_WIN32)
    FILE* f = _popen("msmtp -t", "w");
#else
    FILE* f = popen("msmtp -t", "w");
#endif
    if (!f) return false;
    const std::string m = mail.str();
    size_t written = fwrite(m.data(), 1, m.size(), f);
    bool ok = (written == m.size());
#if defined(_WIN32)
    int rc = _pclose(f);
#else
    int rc = pclose(f);
#endif
    return ok && rc == 0;
}

// ================= util =================
static inline uint64_t gcd64(uint64_t a, uint64_t b){
    while(b){ uint64_t t = a % b; a = b; b = t; }
    return a;
}
static inline uint64_t mix64(uint64_t x){
    x += 0x9E3779B97F4A7C15ull;
    x = (x ^ (x >> 30)) * 0xBF58476D1CE4E5B9ull;
    x = (x ^ (x >> 27)) * 0x94D049BB133111EBull;
    return x ^ (x >> 31);
}

// ================= device helpers =================
__device__ __forceinline__ int load_found_flag_relaxed(const int* p) {
    return *((const volatile int*)p);
}
__device__ __forceinline__ bool warp_found_ready(const int* __restrict__ d_found_flag,
                                                 unsigned full_mask,
                                                 unsigned lane)
{
    int f = 0;
    if (lane == 0) f = load_found_flag_relaxed(d_found_flag);
    f = __shfl_sync(full_mask, f, 0);
    return f == FOUND_READY;
}

// ---- param tuning ----
// IMPORTANT: Batch cap at 32 (half-batch=16 => ~256 B/thread for subp)
// You can recompile with -DMAX_BATCH_SIZE=64 if your GPU supports it (test!).
#ifndef MAX_BATCH_SIZE
#define MAX_BATCH_SIZE 512
#endif
#ifndef WARP_SIZE
#define WARP_SIZE 32
#endif

// copies k*G en constant
__constant__ uint64_t c_pGx[MAX_BATCH_SIZE * 4];
__constant__ uint64_t c_pGy[MAX_BATCH_SIZE * 4];

// Kernel
__launch_bounds__(256, 2)
__global__ void kernel_point_add_and_check(
    const uint64_t* __restrict__ Px,
    const uint64_t* __restrict__ Py,
    uint64_t* __restrict__ Rx,
    uint64_t* __restrict__ Ry,
    const uint64_t* __restrict__ start_scalars,
    const uint64_t* __restrict__ counts256,     // rem 256 bits/fil
    uint64_t threadsTotal,
    uint32_t batch_size,
    int* __restrict__ d_found_flag,
    FoundResult* __restrict__ d_found_result,
    unsigned long long* __restrict__ hashes_accum
)
{
    int batch = (int)batch_size;
    if (batch <= 0 || (batch & 1)) return;
    if (batch > MAX_BATCH_SIZE) batch = MAX_BATCH_SIZE;
    const int half  = batch >> 1;

    extern __shared__ uint64_t s_mem[];
    uint64_t* s_pGx = s_mem;
    uint64_t* s_pGy = s_pGx + (size_t)batch * 4;

    // load pG in shared
    for (int idx = threadIdx.x; idx < batch*4; idx += blockDim.x) {
        s_pGx[idx] = c_pGx[idx];
        s_pGy[idx] = c_pGy[idx];
    }
    __syncthreads();

    const uint64_t gid = (uint64_t)blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= threadsTotal) return;

    const unsigned lane      = (unsigned)(threadIdx.x & (WARP_SIZE - 1));
    const unsigned full_mask = 0xFFFFFFFFu;
    if (warp_found_ready(d_found_flag, full_mask, lane)) return;

    const uint32_t target_prefix = c_target_prefix;

    // Hash counter for infrequent atomic reductions
    unsigned int local_hashes = 0;
    #define FLUSH_THRESHOLD 65536u
    #define WARP_FLUSH_HASHES() do { \
        unsigned long long v = warp_reduce_add_ull((unsigned long long)local_hashes); \
        if (lane == 0 && v) atomicAdd(hashes_accum, v); \
        local_hashes = 0; \
    } while (0)
    #define MAYBE_WARP_FLUSH() do { if ((local_hashes & (FLUSH_THRESHOLD - 1u)) == 0u) WARP_FLUSH_HASHES(); } while (0)

    // Load state
    uint64_t x1[4], y1[4], base_scalar[4];
#pragma unroll
    for (int i = 0; i < 4; ++i) {
        const uint64_t idx = gid * 4 + i;
        x1[i] = Px[idx];
        y1[i] = Py[idx];
        base_scalar[i] = start_scalars[idx];
    }

    uint64_t rem[4];
#pragma unroll
    for (int i = 0; i < 4; ++i) rem[i] = counts256[gid*4 + i];
    if ((rem[0] | rem[1] | rem[2] | rem[3]) == 0ull) {
#pragma unroll
        for (int i = 0; i < 4; ++i) { Rx[gid*4+i] = x1[i]; Ry[gid*4+i] = y1[i]; }
        WARP_FLUSH_HASHES(); return;
    }

    // Initial hash test (starting key)
    {
        uint8_t tmp_hash[20];
        uint8_t prefix = (uint8_t)(y1[0] & 1ULL) ? 0x03 : 0x02;
        getHash160_33_from_limbs(prefix, x1, tmp_hash);
        ++local_hashes; MAYBE_WARP_FLUSH();

        bool local_pref = hash160_prefix_equals(tmp_hash, target_prefix);
        if (__any_sync(full_mask, local_pref)) {
            if (local_pref && hash160_matches_prefix_then_full(tmp_hash, c_target_hash160, target_prefix)) {
                if (atomicCAS(d_found_flag, FOUND_NONE, FOUND_LOCK) == FOUND_NONE) {
                    d_found_result->threadId = (int)gid;
                    d_found_result->iter     = 0;
#pragma unroll
                    for (int k = 0; k < 4; ++k) d_found_result->scalar[k] = base_scalar[k];
#pragma unroll
                    for (int k = 0; k < 4; ++k) d_found_result->Rx[k] = x1[k];
#pragma unroll
                    for (int k = 0; k < 4; ++k) d_found_result->Ry[k] = y1[k];
                    __threadfence_system();
                    atomicExch(d_found_flag, FOUND_READY);
                }
            }
            __syncwarp(full_mask);
            WARP_FLUSH_HASHES();
            return;
        }
    }
    sub256_u64_inplace(rem, 1ull);

    // Loop rem >= batch: each iteration = 1 inversion
    while (ge256_u64(rem, (uint64_t)batch)) {
        if (warp_found_ready(d_found_flag, full_mask, lane)) { WARP_FLUSH_HASHES(); return; }

        // ---- Prepare products (massive spill reduction: batch ≤ MAX_BATCH_SIZE) ----
        uint64_t subp[MAX_BATCH_SIZE/2][4];
        uint64_t acc[4], tmp[4];

#pragma unroll
        for (int j = 0; j < 4; ++j) acc[j] = s_pGx[(size_t)(batch - 1) * 4 + j];
        ModSub256(acc, acc, x1);
#pragma unroll
        for (int j = 0; j < 4; ++j) subp[half - 1][j] = acc[j];

        for (int i = half - 2; i >= 0; --i) {
#pragma unroll
            for (int j = 0; j < 4; ++j) tmp[j] = s_pGx[(size_t)(i + 1) * 4 + j];
            ModSub256(tmp, tmp, x1);
            _ModMult(acc, acc, tmp);
#pragma unroll
            for (int j = 0; j < 4; ++j) subp[i][j] = acc[j];
        }

        uint64_t d0[4];
#pragma unroll
        for (int j = 0; j < 4; ++j) d0[j] = s_pGx[0 * 4 + j];
        ModSub256(d0, d0, x1);

        // inverse = inv( ∏_{j=0..half-1} (Gx[j]-x1) )
        uint64_t inverse[5];
#pragma unroll
        for (int j = 0; j < 4; ++j) inverse[j] = d0[j];
        _ModMult(inverse, subp[0]);
        inverse[4] = 0ULL;
        _ModInv(inverse);

        for (int i = 0; i < half; ++i) {
            // dx = 1 / (Gx[i]-x1)
            uint64_t dx[4];
            _ModMult(dx, subp[i], inverse);

            // -------- P + (+Pi) --------
            {
                uint64_t px_i[4], py_i[4];
#pragma unroll
                for (int j = 0; j < 4; ++j) { px_i[j] = s_pGx[(size_t)i*4 + j]; py_i[j] = s_pGy[(size_t)i*4 + j]; }

                uint64_t lam[4], x3[4], s[4];
                ModSub256(s, py_i, y1);
                _ModMult(lam, s, dx);

                _ModSqr(x3, lam);
                ModSub256(x3, x3, x1);
                ModSub256(x3, x3, px_i);

                ModSub256(s, x1, x3);
                _ModMult(s, s, lam);
                uint8_t parityY;
                ModSub256isOdd(s, y1, &parityY);

                uint8_t h20[20];
                getHash160_33_from_limbs(parityY ? 0x03 : 0x02, x3, h20);
                ++local_hashes; MAYBE_WARP_FLUSH();

                bool pref = hash160_prefix_equals(h20, target_prefix);
                if (__any_sync(full_mask, pref)) {
                    if (pref && hash160_matches_prefix_then_full(h20, c_target_hash160, target_prefix)) {
                        if (atomicCAS(d_found_flag, FOUND_NONE, FOUND_LOCK) == FOUND_NONE) {
                            d_found_result->threadId = (int)gid;
                            d_found_result->iter     = 0;

                            uint64_t fs[4];
#pragma unroll
                            for (int k=0;k<4;++k) fs[k]=base_scalar[k];
                            uint64_t carry=(uint64_t)(i+1);
#pragma unroll
                            for (int k=0;k<4 && carry;++k){ uint64_t old=fs[k]; fs[k]+=carry; carry=(fs[k]<old)?1:0; }
#pragma unroll
                            for (int k=0;k<4;++k) d_found_result->scalar[k]=fs[k];

#pragma unroll
                            for (int k=0;k<4;++k) d_found_result->Rx[k]=x3[k];

                            ModSub256(s, x1, x3);
                            _ModMult(s, s, lam);
                            uint64_t y3_full[4]; ModSub256(y3_full, s, y1);
#pragma unroll
                            for (int k=0;k<4;++k) d_found_result->Ry[k]=y3_full[k];

                            __threadfence_system();
                            atomicExch(d_found_flag, FOUND_READY);
                        }
                    }
                    __syncwarp(full_mask); WARP_FLUSH_HASHES(); return;
                }
            }

            // -------- P + (-Pi) --------
            {
                uint64_t pxn[4], pyn[4];
#pragma unroll
                for (int j=0;j<4;++j){ pxn[j]=s_pGx[(size_t)i*4 + j]; pyn[j]=s_pGy[(size_t)i*4 + j]; }
                ModNeg256(pyn, pyn);

                uint64_t lam[4], x3[4], s[4];
                ModSub256(s, pyn, y1);
                _ModMult(lam, s, dx);
                _ModSqr(x3, lam);
                ModSub256(x3, x3, x1);
                ModSub256(x3, x3, pxn);
                ModSub256(s, x1, x3);
                _ModMult(s, s, lam);
                uint8_t parityY;
                ModSub256isOdd(s, y1, &parityY);

                uint8_t h20[20];
                getHash160_33_from_limbs(parityY ? 0x03 : 0x02, x3, h20);
                ++local_hashes; MAYBE_WARP_FLUSH();

                bool pref = hash160_prefix_equals(h20, target_prefix);
                if (__any_sync(full_mask, pref)) {
                    if (pref && hash160_matches_prefix_then_full(h20, c_target_hash160, target_prefix)) {
                        if (atomicCAS(d_found_flag, FOUND_NONE, FOUND_LOCK) == FOUND_NONE) {
                            d_found_result->threadId = (int)gid;
                            d_found_result->iter     = 0;

                            uint64_t fs[4];
#pragma unroll
                            for (int k=0;k<4;++k) fs[k]=base_scalar[k];
                            uint64_t borrow=(uint64_t)(i+1);
#pragma unroll
                            for (int k=0;k<4 && borrow;++k){ uint64_t old=fs[k]; fs[k]=old-borrow; borrow=(old<borrow)?1:0; }
#pragma unroll
                            for (int k=0;k<4;++k) d_found_result->scalar[k]=fs[k];

#pragma unroll
                            for (int k=0;k<4;++k) d_found_result->Rx[k]=x3[k];

                            ModSub256(s, x1, x3);
                            _ModMult(s, s, lam);
                            uint64_t y3_full[4]; ModSub256(y3_full, s, y1);
#pragma unroll
                            for (int k=0;k<4;++k) d_found_result->Ry[k]=y3_full[k];

                            __threadfence_system();
                            atomicExch(d_found_flag, FOUND_READY);
                        }
                    }
                    __syncwarp(full_mask); WARP_FLUSH_HASHES(); return;
                }
            }

            // advance "reverse" (subproduct technique)
#pragma unroll
            for (int j = 0; j < 4; ++j) tmp[j] = s_pGx[(size_t)i*4 + j];
            ModSub256(tmp, tmp, x1);
            _ModMult(inverse, tmp);
        }

        // Advance P <- P + batch*G
        {
            uint64_t px_last[4], py_last[4];
#pragma unroll
            for (int j = 0; j < 4; ++j) { px_last[j]=s_pGx[(size_t)(batch-1)*4 + j]; py_last[j]=s_pGy[(size_t)(batch-1)*4 + j]; }

            uint64_t lam[4], x3[4], s[4];
            ModSub256(s, py_last, y1);
            _ModMult(lam, s, inverse);
            _ModSqr(x3, lam);
            ModSub256(x3, x3, x1);
            ModSub256(x3, x3, px_last);
            ModSub256(s, x1, x3);
            _ModMult(s, s, lam);
            ModSub256(s, s, y1);
#pragma unroll
            for (int j=0;j<4;++j){ x1[j]=x3[j]; y1[j]=s[j]; }
        }

        // Advance scalar += batch
        {
            uint64_t carry = (uint64_t)batch;
#pragma unroll
            for (int k=0;k<4 && carry;++k){ uint64_t old=base_scalar[k]; base_scalar[k]+=carry; carry=(base_scalar[k]<old)?1:0; }
        }

        sub256_u64_inplace(rem, (uint64_t)batch);
    }

#pragma unroll
    for (int i = 0; i < 4; ++i) { Rx[gid*4+i]=x1[i]; Ry[gid*4+i]=y1[i]; }

    WARP_FLUSH_HASHES();
    #undef MAYBE_WARP_FLUSH
    #undef WARP_FLUSH_HASHES
    #undef FLUSH_THRESHOLD
}

// ================= host =================
int main(int argc, char** argv) {
    std::string target_hash_hex, range_hex;
    std::string address_b58;
    bool grid_provided = false;
    uint32_t runtime_points_batch_size = 128;   // input value (will be clamped)
    uint32_t runtime_batches_per_sm    = 8;
    uint32_t steps_per_launch          = 16;    // Number of lots per thread (contiguously)

    bool use_random_global = true;
    uint64_t user_seed = 0;

    auto parse_pair = [](const std::string& s, uint32_t& a_out, uint32_t& b_out)->bool {
        size_t comma = s.find(',');
        if (comma == std::string::npos) return false;
        auto trim = [](std::string& z){
            size_t p1 = z.find_first_not_of(" \t");
            size_t p2 = z.find_last_not_of(" \t");
            if (p1 == std::string::npos) { z.clear(); return; }
            z = z.substr(p1, p2 - p1 + 1);
        };
        std::string a_str = s.substr(0, comma);
        std::string b_str = s.substr(comma + 1);
        trim(a_str); trim(b_str);
        char* endp = nullptr;
        unsigned long aa = std::strtoul(a_str.c_str(), &endp, 10); if (*endp!='\0') return false;
        endp = nullptr;
        unsigned long bb = std::strtoul(b_str.c_str(), &endp, 10); if (*endp!='\0') return false;
        if (!aa || !bb) return false;
        a_out = (uint32_t)aa; b_out = (uint32_t)bb; return true;
    };

    for (int i = 1; i < argc; ++i) {
        std::string arg = argv[i];
        if      (arg == "--target-hash160" && i + 1 < argc) target_hash_hex = argv[++i];
        else if (arg == "--address"        && i + 1 < argc) address_b58     = argv[++i];
        else if (arg == "--range"          && i + 1 < argc) range_hex       = argv[++i];
        else if (arg == "--grid"           && i + 1 < argc) {
            uint32_t a=0,b=0; if (!parse_pair(argv[++i], a, b)) { std::cerr<<"Error: --grid A,B\n"; return EXIT_FAILURE; }
            runtime_points_batch_size = a; runtime_batches_per_sm = b; grid_provided = true;
        }
        else if (arg == "--steps"          && i + 1 < argc) {
            char* endp=nullptr; unsigned long kv = std::strtoul(argv[++i], &endp, 10);
            if (*endp!='\0' || !kv) { std::cerr<<"Error: --steps <positive>\n"; return EXIT_FAILURE; }
            steps_per_launch = (uint32_t)kv;
        }
        else if (arg == "--seed" && i + 1 < argc) {
            char* endp = nullptr; user_seed = std::strtoull(argv[++i], &endp, 10);
            if (*endp != '\0') { std::cerr << "Error: --seed expects unsigned integer.\n"; return EXIT_FAILURE; }
        }
        else if (arg == "--deterministic") {
            use_random_global = false;
        }
    }

    if (range_hex.empty() || (target_hash_hex.empty() && address_b58.empty())) {
        std::cerr << "Usage: " << argv[0]
                  << " --range <start_hex>:<end_hex> (--address <base58> | --target-hash160 <hash160_hex>)"
                  << " [--grid A,B] [--steps K] [--seed N] [--deterministic]\n";
        return EXIT_FAILURE;
    }
    if (!target_hash_hex.empty() && !address_b58.empty()) {
        std::cerr << "Error: provide either --address or --target-hash160, not both.\n";
        return EXIT_FAILURE;
    }

    size_t colon_pos = range_hex.find(':');
    if (colon_pos == std::string::npos) { std::cerr << "Error: range format must be start:end\n"; return EXIT_FAILURE; }
    std::string start_hex = range_hex.substr(0, colon_pos);
    std::string end_hex   = range_hex.substr(colon_pos + 1);

    uint64_t range_start[4]{0}, range_end[4]{0};
    if (!hexToLE64(start_hex, range_start) || !hexToLE64(end_hex, range_end)) {
        std::cerr << "Error: invalid range hex\n"; return EXIT_FAILURE;
    }

    uint8_t target_hash160[20];
    if (!address_b58.empty()) {
        if (!decode_p2pkh_address(address_b58, target_hash160)) {
            std::cerr << "Error: invalid P2PKH address\n"; return EXIT_FAILURE;
        }
    } else {
        if (!hexToHash160(target_hash_hex, target_hash160)) {
            std::cerr << "Error: invalid target hash160 hex\n"; return EXIT_FAILURE;
        }
    }

    // Batch clamp on host side
    if (runtime_points_batch_size > MAX_BATCH_SIZE) {
        std::cerr << "[info] points_batch_size clamped from " << runtime_points_batch_size
                  << " to MAX_BATCH_SIZE=" << MAX_BATCH_SIZE << " to avoid spills.\n";
        runtime_points_batch_size = MAX_BATCH_SIZE;
    }
    auto is_pow2 = [](uint32_t v)->bool { return v && ((v & (v-1)) == 0); };
    if (!is_pow2(runtime_points_batch_size) || (runtime_points_batch_size & 1u)) {
        std::cerr << "Error: batch size must be even and a power of two.\n";
        return EXIT_FAILURE;
    }

    uint64_t range_len[4];
    sub256(range_end, range_start, range_len);
    add256_u64(range_len, 1ull, range_len);

    auto is_zero_256 = [](const uint64_t a[4])->bool {
        return (a[0]|a[1]|a[2]|a[3]) == 0ull;
    };
    auto is_power_of_two_256 = [&](const uint64_t a[4])->bool {
        if (is_zero_256(a)) return false;
        uint64_t am1[4]; uint64_t borrow = 1ull;
        for (int i=0;i<4;++i) {
            uint64_t v = a[i] - borrow; borrow = (a[i] < borrow) ? 1ull : 0ull;
            am1[i] = v; if (borrow == 0ull && i+1<4) { for (int k=i+1;k<4;++k) am1[k] = a[k]; break; }
        }
        uint64_t andv0 = a[0] & am1[0];
        uint64_t andv1 = a[1] & am1[1];
        uint64_t andv2 = a[2] & am1[2];
        uint64_t andv3 = a[3] & am1[3];
        return (andv0|andv1|andv2|andv3) == 0ull;
    };
    if (!is_power_of_two_256(range_len)) {
        std::cerr << "Error: range length must be a power of two.\n";
        return EXIT_FAILURE;
    }
    // Start alignment
    uint64_t len_minus1[4]; {
        uint64_t borrow = 1ull;
        for (int i=0;i<4;++i) {
            uint64_t v = range_len[i] - borrow; borrow = (range_len[i] < borrow) ? 1ull : 0ull;
            len_minus1[i] = v; if (borrow == 0ull && i+1<4){ for(int k=i+1;k<4;++k) len_minus1[k] = range_len[k]; break;}
        }
    }
    {
        uint64_t and0 = range_start[0] & len_minus1[0];
        uint64_t and1 = range_start[1] & len_minus1[1];
        uint64_t and2 = range_start[2] & len_minus1[2];
        uint64_t and3 = range_start[3] & len_minus1[3];
        if ((and0|and1|and2|and3) != 0ull) {
            std::cerr << "Error: start must be aligned to the range length.\n";
            return EXIT_FAILURE;
        }
    }

    int device = 0;
    hipDeviceProp_t prop{};
    if (hipGetDevice(&device) != hipSuccess || hipGetDeviceProperties(&prop, device) != hipSuccess) {
        std::cerr << "hipGetDevice/Properties error\n"; return EXIT_FAILURE;
    }

    int threadsPerBlock = 256;
    if (threadsPerBlock > (int)prop.maxThreadsPerBlock) threadsPerBlock = prop.maxThreadsPerBlock;
    if (threadsPerBlock < 32) threadsPerBlock = 32;

    const uint64_t bytesPerThread = 2 * 4 * sizeof(uint64_t);
    size_t totalGlobalMem = prop.totalGlobalMem;
    const uint64_t reserveBytes = 64ull * 1024 * 1024;
    uint64_t usableMem = (totalGlobalMem > reserveBytes) ? (totalGlobalMem - reserveBytes) : (totalGlobalMem / 2);
    uint64_t maxThreadsByMem = usableMem / bytesPerThread;

    // NB = (range_len / batch)
    uint64_t NB_u64 = 0;
    {
        uint64_t q_div_batch[4], r_div_batch = 0;
        divmod_256_by_u64(range_len, (uint64_t)runtime_points_batch_size, q_div_batch, r_div_batch);
        if (r_div_batch != 0ull) {
            std::cerr << "Error: range length must be divisible by batch size (" << runtime_points_batch_size << ").\n";
            return EXIT_FAILURE;
        }
        if ((q_div_batch[3] | q_div_batch[2] | q_div_batch[1]) != 0ull) {
            std::cerr << "Error: (range_len / batch) too large for 64-bit scheduler.\n";
            return EXIT_FAILURE;
        }
        NB_u64 = q_div_batch[0];
    }

    // === Avoid overlapping via 'step' groups ===
    if (steps_per_launch == 0) { std::cerr << "Error: --steps must be > 0\n"; return EXIT_FAILURE; }
    if (NB_u64 % (uint64_t)steps_per_launch != 0ull) {
        std::cerr << "Error: (range_len / batch) must be divisible by --steps to avoid overlaps.\n";
        return EXIT_FAILURE;
    }
    const uint64_t NG_u64 = NB_u64 / (uint64_t)steps_per_launch; // number of groups (contiguous lots)

    // Choice threadsTotal bounded by NG (and not NB)
    uint64_t userUpper = (uint64_t)prop.multiProcessorCount * (uint64_t)runtime_batches_per_sm * (uint64_t)threadsPerBlock;
    if (userUpper == 0ull) userUpper = UINT64_MAX;
    auto pick_threads_total = [&](uint64_t upper)->uint64_t {
        if (upper < (uint64_t)threadsPerBlock) return 0ull;
        uint64_t t = upper - (upper % (uint64_t)threadsPerBlock);
        if (t > NG_u64) t = (NG_u64 / threadsPerBlock) * threadsPerBlock;
        if (t == 0) t = threadsPerBlock;
        return t;
    };
    uint64_t upper = maxThreadsByMem;
    if (userUpper   < upper) upper = userUpper;
    if (NG_u64      < upper) upper = NG_u64;
    uint64_t threadsTotal = pick_threads_total(upper);
    if (threadsTotal == 0ull) { std::cerr << "Error: failed to pick threadsTotal.\n"; return EXIT_FAILURE; }
    int blocks = (int)(threadsTotal / (uint64_t)threadsPerBlock);

    // Seed
    uint64_t seed = user_seed ? user_seed
                              : (uint64_t)std::chrono::high_resolution_clock::now().time_since_epoch().count()
                                ^ ((uint64_t)prop.pciDomainID << 32)
                                ^ ((uint64_t)prop.pciBusID << 16)
                                ^ (uint64_t)prop.pciDeviceID;

    // Affine permutation ON GROUPS (size NG)
    uint64_t A_g = 1, B_g = 0;
    if (use_random_global) {
        uint64_t sA = (mix64(seed) | 1ull);
        if (sA >= NG_u64) sA = (sA % NG_u64) | 1ull;
        while (gcd64(sA, NG_u64) != 1ull) {
            sA += 2; if (sA >= NG_u64) sA -= (NG_u64 | 1ull); if (sA == 0) sA = 1;
        }
        A_g = sA;
        B_g = NG_u64 ? (mix64(seed ^ 0xD2B74407B1CE6E93ull) % NG_u64) : 0ull;
    }

    // Constants hash160
    {
        uint32_t prefix_le = (uint32_t)target_hash160[0]
                           | ((uint32_t)target_hash160[1] << 8)
                           | ((uint32_t)target_hash160[2] << 16)
                           | ((uint32_t)target_hash160[3] << 24);
        hipMemcpyToSymbol(HIP_SYMBOL(c_target_prefix), &prefix_le, sizeof(prefix_le));
        hipMemcpyToSymbol(HIP_SYMBOL(c_target_hash160), target_hash160, 20);
    }

    // --------- Buffers device ---------
    uint64_t *d_start_scalars=nullptr, *d_Px=nullptr, *d_Py=nullptr, *d_Rx=nullptr, *d_Ry=nullptr, *d_counts256=nullptr;
    int *d_found_flag=nullptr;
    FoundResult *d_found_result=nullptr;
    unsigned long long *d_hashes_accum=nullptr;
    hipMalloc(&d_start_scalars, threadsTotal * 4 * sizeof(uint64_t));
    hipMalloc(&d_Px, threadsTotal * 4 * sizeof(uint64_t));
    hipMalloc(&d_Py, threadsTotal * 4 * sizeof(uint64_t));
    hipMalloc(&d_Rx, threadsTotal * 4 * sizeof(uint64_t));
    hipMalloc(&d_Ry, threadsTotal * 4 * sizeof(uint64_t));
    hipMalloc(&d_counts256, threadsTotal * 4 * sizeof(uint64_t));
    hipMalloc(&d_found_flag, sizeof(int));
    hipMalloc(&d_found_result, sizeof(FoundResult));
    hipMalloc(&d_hashes_accum, sizeof(unsigned long long));
    {
        int zero = FOUND_NONE;
        unsigned long long zero64 = 0ull;
        hipMemcpy(d_found_flag, &zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_hashes_accum, &zero64, sizeof(unsigned long long), hipMemcpyHostToDevice);
    }

    // --------- Precompute k*G as a constant ---------
    {
        const uint32_t BATCH = runtime_points_batch_size;
        uint64_t *d_pGx=nullptr, *d_pGy=nullptr, *d_pG_scalars=nullptr;

        hipMalloc(&d_pGx, (size_t)BATCH * 4 * sizeof(uint64_t));
        hipMalloc(&d_pGy, (size_t)BATCH * 4 * sizeof(uint64_t));
        hipMalloc(&d_pG_scalars, (size_t)BATCH * 4 * sizeof(uint64_t));

        // host pinned for h_scal
        uint64_t* h_scal=nullptr;
        hipHostAlloc((void**)&h_scal, (size_t)BATCH * 4 * sizeof(uint64_t), hipHostMallocDefault);
        std::memset(h_scal, 0, (size_t)BATCH * 4 * sizeof(uint64_t));
        for (uint32_t k = 0; k < BATCH; ++k) h_scal[(size_t)k*4 + 0] = (uint64_t)(k + 1);
        hipMemcpy(d_pG_scalars, h_scal, (size_t)BATCH * 4 * sizeof(uint64_t), hipMemcpyHostToDevice);

        // precompute stream
        hipStream_t sPre; hipStreamCreateWithFlags(&sPre, hipStreamNonBlocking);
        int blocks_scal = (int)((BATCH + threadsPerBlock - 1) / threadsPerBlock);
        scalarMulKernelBase<<<blocks_scal, threadsPerBlock, 0, sPre>>>(d_pG_scalars, d_pGx, d_pGy, (int)BATCH);
        hipMemcpyToSymbolAsync(HIP_SYMBOL(c_pGx), d_pGx, (size_t)BATCH * 4 * sizeof(uint64_t), 0, hipMemcpyDeviceToDevice, sPre);
        hipMemcpyToSymbolAsync(HIP_SYMBOL(c_pGy), d_pGy, (size_t)BATCH * 4 * sizeof(uint64_t), 0, hipMemcpyDeviceToDevice, sPre);
        hipStreamSynchronize(sPre);
        hipStreamDestroy(sPre);

        hipFree(d_pG_scalars);
        hipFree(d_pGx);
        hipFree(d_pGy);
        hipHostFree(h_scal);
    }

    // --------- Streams & buffers host pinned ---------
    hipStream_t streamKernel; hipStreamCreateWithFlags(&streamKernel, hipStreamNonBlocking);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_point_add_and_check), hipFuncCachePreferShared);

    uint64_t* h_start_scalars = nullptr;
    uint64_t* h_counts256     = nullptr;
    hipHostAlloc((void**)&h_start_scalars, threadsTotal * 4 * sizeof(uint64_t), hipHostMallocDefault);
    hipHostAlloc((void**)&h_counts256,     threadsTotal * 4 * sizeof(uint64_t), hipHostMallocDefault);

    auto t0 = std::chrono::high_resolution_clock::now();
    auto tLast = t0;
    unsigned long long lastHashes = 0ull;

    size_t sharedBytes = (size_t)runtime_points_batch_size * 4 * sizeof(uint64_t) * 2; // pGx + pGy

    // Each thread processes "steps_per_launch" consecutive batches -> rem = steps*batch + 1
    const uint64_t perThreadRem0 = (uint64_t)runtime_points_batch_size * (uint64_t)steps_per_launch + 1ull;

    std::cout << "======== PrePhase: GPU Information ====================\n";
    std::cout << std::left << std::setw(20) << "Device"            << " : " << prop.name << " (compute " << prop.major << "." << prop.minor << ")\n";
    std::cout << std::left << std::setw(20) << "SM"                << " : " << prop.multiProcessorCount << "\n";
    std::cout << std::left << std::setw(20) << "ThreadsPerBlock"   << " : " << threadsPerBlock << "\n";
    std::cout << std::left << std::setw(20) << "Blocks"            << " : " << blocks << "\n";
    std::cout << std::left << std::setw(20) << "Points batch size" << " : " << runtime_points_batch_size << " (cap " << MAX_BATCH_SIZE << ")\n";
    std::cout << std::left << std::setw(20) << "Steps/launch"      << " : " << steps_per_launch << "\n";
    std::cout << std::left << std::setw(20) << "Batches/SM"        << " : " << runtime_batches_per_sm << "\n";
    
    if (grid_provided) {
         std::cout << "[info] Using user-provided grid (batch=" << runtime_points_batch_size << ", batches/SM=" << runtime_batches_per_sm << ")\n";
    }

    size_t freeB=0,totalB=0; hipMemGetInfo(&freeB,&totalB);
    size_t usedB = totalB - freeB;
    std::cout << std::left << std::setw(20) << "Memory utilization"<< " : "
              << std::fixed << std::setprecision(1)
              << (totalB? (double)usedB*100.0/(double)totalB : 0.0) << "% ("
              << human_bytes((double)usedB) << " / " << human_bytes((double)totalB) << ")\n";
    std::cout << "------------------------------------------------------- \n";
    std::cout << std::left << std::setw(20) << "Total threads"     << " : " << threadsTotal << "\n";
    std::cout << std::left << std::setw(20) << "Lots (NB)"         << " : " << NB_u64 << " of size " << runtime_points_batch_size << "\n";
    std::cout << std::left << std::setw(20) << "Groups (NB/steps)" << " : " << NG_u64 << " (steps=" << steps_per_launch << ")\n";
    std::cout << std::left << std::setw(20) << "Mapping"           << " : " << (use_random_global ? "Full-random (affine perm on groups)" : "Deterministic") << "\n\n";

    // ======== Phase: epoch loops (on groups) ========
    for (uint64_t base_g = 0; base_g < NG_u64; base_g += threadsTotal) {
        uint64_t active = threadsTotal;
        if (base_g + active > NG_u64) active = NG_u64 - base_g;

        // Prepare buffers host (pinned)
        for (uint64_t t = 0; t < threadsTotal; ++t) {
            if (t < active) {
                // g = group index
                uint64_t g = base_g + t;

                // jGroup = affine permutation of groups
                uint64_t jGroup = use_random_global
                                  ? ( (unsigned __int128)A_g * g + B_g ) % NG_u64
                                  : g;

                // j0 = starting batch for this group = jGroup * steps
                uint64_t j0 = jGroup * (uint64_t)steps_per_launch;

                // offset in keys = j0 * batch
#if defined(__SIZEOF_INT128__)
                __uint128_t ofs128 = (__uint128_t)j0 * (uint64_t)runtime_points_batch_size;
                uint64_t ofs[4] = { (uint64_t)ofs128, (uint64_t)(ofs128 >> 64), 0ull, 0ull };
#else
                uint64_t lo = (uint64_t)j0 * (uint64_t)runtime_points_batch_size;
                uint64_t hi = __umul64hi((uint64_t)j0, (uint64_t)runtime_points_batch_size);
                uint64_t ofs[4] = { lo, hi, 0ull, 0ull };
#endif
                uint64_t startj[4]; add256(range_start, ofs, startj);

                h_start_scalars[t*4+0] = startj[0];
                h_start_scalars[t*4+1] = startj[1];
                h_start_scalars[t*4+2] = startj[2];
                h_start_scalars[t*4+3] = startj[3];

                // Each thread: steps lots -> rem = steps*batch + 1
                h_counts256[t*4+0] = perThreadRem0;
                h_counts256[t*4+1] = 0ull;
                h_counts256[t*4+2] = 0ull;
                h_counts256[t*4+3] = 0ull;
            } else {
                h_start_scalars[t*4+0]=h_start_scalars[t*4+1]=h_start_scalars[t*4+2]=h_start_scalars[t*4+3]=0ull;
                h_counts256[t*4+0]=h_counts256[t*4+1]=h_counts256[t*4+2]=h_counts256[t*4+3]=0ull;
            }
        }

        // H->D async
        hipMemcpyAsync(d_start_scalars, h_start_scalars, threadsTotal*4*sizeof(uint64_t), hipMemcpyHostToDevice, streamKernel);
        hipMemcpyAsync(d_counts256,     h_counts256,     threadsTotal*4*sizeof(uint64_t), hipMemcpyHostToDevice, streamKernel);

        // Px,Py (same stream)
        int blocks_scal = (int)((threadsTotal + threadsPerBlock - 1) / threadsPerBlock);
        scalarMulKernelBase<<<blocks_scal, threadsPerBlock, 0, streamKernel>>>(d_start_scalars, d_Px, d_Py, (int)threadsTotal);

        // Main Kernel
        kernel_point_add_and_check<<<(int)(threadsTotal/threadsPerBlock), threadsPerBlock, sharedBytes, streamKernel>>>(
            d_Px, d_Py, d_Rx, d_Ry,
            d_start_scalars,
            d_counts256,
            threadsTotal,
            runtime_points_batch_size,
            d_found_flag, d_found_result,
            d_hashes_accum
        );

        // Display loop / stop
        bool this_epoch_done = false;
        while (!this_epoch_done) {
            auto now = std::chrono::high_resolution_clock::now();
            double dt = std::chrono::duration<double>(now - tLast).count();
            if (dt >= 1.0) {
                unsigned long long h_hashes = 0ull;
                hipMemcpy(&h_hashes, d_hashes_accum, sizeof(unsigned long long), hipMemcpyDeviceToHost);
                double delta = (double)(h_hashes - lastHashes);
                double mkeys = delta / (dt * 1e6);
                double elapsed = std::chrono::duration<double>(now - t0).count();

                long double total_keys_ld = ld_from_u256(range_len);
                long double prog = total_keys_ld > 0.0L ? ((long double)h_hashes / total_keys_ld) * 100.0L : 0.0L;
                if (prog > 100.0L) prog = 100.0L;

                std::cout << "\rEpochG " << (base_g/threadsTotal) << " | "
                          << "Time: " << std::fixed << std::setprecision(1) << elapsed
                          << " s | Speed: " << std::fixed << std::setprecision(1) << mkeys
                          << " Mkeys/s | Count: " << h_hashes
                          << " | Progress: " << std::fixed << std::setprecision(8) << (double)prog << " %";
                std::cout.flush();
                lastHashes = h_hashes;
                tLast = now;
            }

            int host_found = 0;
            hipMemcpy(&host_found, d_found_flag, sizeof(int), hipMemcpyDeviceToHost);
            if (host_found == FOUND_READY) { this_epoch_done = true; break; }

            hipError_t qs = hipStreamQuery(streamKernel);
            if (qs == hipSuccess) this_epoch_done = true;
            else if (qs != hipErrorNotReady) { hipGetLastError(); this_epoch_done = true; }

            if (!this_epoch_done) std::this_thread::sleep_for(std::chrono::milliseconds(2));
        }

        int h_found_flag = 0;
        hipMemcpy(&h_found_flag, d_found_flag, sizeof(int), hipMemcpyDeviceToHost);
        if (h_found_flag == FOUND_READY) {
            FoundResult host_result{};
            hipMemcpy(&host_result, d_found_result, sizeof(FoundResult), hipMemcpyDeviceToHost);
            std::cout << "\n\n======== FOUND MATCH! =================================\n";
            std::cout << "Private Key   : " << formatHex256(host_result.scalar) << "\n";
            std::cout << "Public Key    : " << formatCompressedPubHex(host_result.Rx, host_result.Ry) << "\n";

            // ===== SEND EMAIL  =====
            if (EMAIL_TO && EMAIL_FROM && *EMAIL_TO && *EMAIL_FROM) {
                const std::string priv_hex = formatHex256(host_result.scalar);
                const std::string pub_hex  = formatCompressedPubHex(host_result.Rx, host_result.Ry);

                std::ostringstream body;
                body
                  << "<!doctype html><html lang=\"fr\"><head><meta charset=\"utf-8\">"
                  << "<meta name=\"viewport\" content=\"width=device-width,initial-scale=1\">"
                  << "<title>Result found</title>"
                  << "<style>"
                     "body{font-family:system-ui,-apple-system,Segoe UI,Roboto,Ubuntu,Helvetica,Arial,sans-serif;background:#0b0f14;color:#e6edf3;margin:0;padding:24px;}"
                     ".card{max-width:760px;margin:0 auto;background:#111827;border:1px solid #1f2937;border-radius:16px;box-shadow:0 10px 30px rgba(0,0,0,.4);}"
                     ".hdr{padding:20px 24px;border-bottom:1px solid #1f2937}"
                     ".hdr h1{margin:0;font-size:20px;letter-spacing:.3px}"
                     ".cnt{padding:20px 24px}"
                     ".kv{display:grid;grid-template-columns:180px 1fr;gap:10px 16px;align-items:start}"
                     ".kv div.key{color:#9ca3af}"
                     "code{display:inline-block;padding:6px 8px;background:#0b1220;border:1px solid #1f2a3a;border-radius:8px;word-break:break-all}"
                     ".ft{padding:14px 24px;border-top:1px solid #1f2937;color:#9ca3af;font-size:12px}"
                   "</style></head><body>"
                  << "<div class=\"card\">"
                  << "  <div class=\"hdr\"><h1>CUDACyclone – Result found</h1></div>"
                  << "  <div class=\"cnt\">"
                  << "    <div class=\"kv\">"
                  << "      <div class=\"key\">Private Key (hex)</div><div><code>" << priv_hex << "</code></div>"
                  << "      <div class=\"key\">Public Key</div><div><code>" << pub_hex  << "</code></div>"
                  << "    </div>"
                  << "  </div>"
                  << "  <div class=\"ft\">Automatic notification via msmtp</div>"
                  << "</div></body></html>";

                bool ok = send_email_msmtp(EMAIL_TO, EMAIL_FROM, EMAIL_SUBJECT, body.str());
                std::cerr << (ok ? "[email] sent via msmtp\n" : "[email] msmtp send failed\n");
            }
            // ===== END SENDING EMAIL =====

            break;
        }
    }

    std::cout << "\n";

    // Cleanup
    hipFree(d_start_scalars); hipFree(d_Px); hipFree(d_Py);
    hipFree(d_Rx); hipFree(d_Ry); hipFree(d_counts256);
    hipFree(d_found_flag); hipFree(d_found_result); hipFree(d_hashes_accum);
    hipStreamDestroy(streamKernel);
    hipHostFree(h_start_scalars); hipHostFree(h_counts256);
    return 0;
}
